#include "hip/hip_runtime.h"
//
// Assignment 1: ParallelSine
// CSCI 415: Networking and Parallel Computation
// Spring 2017
// Name(s): 
//
// Sine implementation derived from slides here: http://15418.courses.cs.cmu.edu/spring2016/lecture/basicarch


// standard imports
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>

// problem size (vector length) N
static const int N = 12345678;

// Number of terms to use when approximating sine
static const int TERMS = 6;

// kernel function (CPU - Do not modify)
void sine_serial(float *input, float *output)
{
  int i;

  for (i=0; i<N; i++) {
      float value = input[i]; 
      float numer = input[i] * input[i] * input[i]; 
      int denom = 6; // 3! 
      int sign = -1; 
      for (int j=1; j<=TERMS;j++) 
      { 
         value += sign * numer / denom; 
         numer *= input[i] * input[i]; 
         denom *= (2*j+2) * (2*j+3); 
         sign *= -1; 
      } 
      output[i] = value; 
    }
}


// kernel function (CUDA device)
// TODO: Implement your graphics kernel here. See assignment instructions for method information


__global__ void sine_parallel(float *input, float *output)
{

	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	if(thread_id < N){
	
	float value = input[thread_id];
	float numer = input[thread_id] * input[thread_id] * input[thread_id];
	int denom = 6; // 3!
	int sign = -1;

	for(int j = 1; j<= TERMS; j++)
{
		value += sign * numer / denom;
		numer *= input[thread_id] * input[thread_id];
		denom *= (2*j+2) * (2*j+3);
		sign *= -1;
}
	output[thread_id] = value; 
}
}
// BEGIN: timing and error checking routines (do not modify)

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}

void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

// END: timing and error checking routines (do not modify)



int main (int argc, char **argv)
{
  //BEGIN: CPU implementation (do not modify)
  float *h_cpu_result = (float*)malloc(N*sizeof(float));
  float *h_input = (float*)malloc(N*sizeof(float));
  //Initialize data on CPU
  int i;
  for (i=0; i<N; i++)
  {
    h_input[i] = 0.1f * i;
  }

  //Execute and time the CPU version
  long long CPU_start_time = start_timer();
  sine_serial(h_input, h_cpu_result);
  long long CPU_time = stop_timer(CPU_start_time, "\nCPU Run Time");
  //END: CPU implementation (do not modify)


  //TODO: Prepare and run your kernel, make sure to copy your results back into h_gpu_result and display your timing results
  float *h_gpu_result = (float*)malloc(N*sizeof(float));
  
  //memory pointers for GPU
  float *g_input;

  float *g_output;

  //Start time for the GPU
  long long GPU_start_time = start_timer();

  //memory allocation for the GPU
  long long memoryAllocate_GPU_start = start_timer();
  hipMalloc((void **) &g_output, N*sizeof(float));

  hipMalloc((void **) &g_input, N*sizeof(float));
  long long memoryAllocate_GPU_time = stop_timer(memoryAllocate_GPU_start, "\n GPU - Copy Memory to Device");


  //transer info to the GPU
  long long memory_to_GPU_start = start_timer();
  hipMemcpy(g_input, h_input, N*sizeof(float), hipMemcpyHostToDevice);
  long long memory_to_GPU_time = stop_timer(memory_to_GPU_start, "\nMemory to GPU time");

  //start kernel with time
  long long kernel_start_time = start_timer();
  sine_parallel<<<12057,1024>>>( g_input,g_output);
  long long GPU_time = stop_timer(kernel_start_time, "\nKernel Running Time");

  //results back to the CPU
  long long memory_to_host_time_start = start_timer();
  hipMemcpy(h_gpu_result, g_output,N*sizeof(float), hipMemcpyDeviceToHost);
  long long memory_to_host_time = stop_timer(memory_to_host_time_start,"\nGPU Memory to Host time");
  


  //total time for GPU process
  long long total_runtime = stop_timer(GPU_start_time, "\nTotal Runtime GPU");

  //Memory Clean
  hipFree(g_input);
  hipFree(g_output);


  // Checking to make sure the CPU and GPU results match - Do not modify
  int errorCount = 0;
  for (i=0; i<N; i++)
  {
    if (abs(h_cpu_result[i]-h_gpu_result[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
   
  else
    printf("Result comparison passed.\n");
 
  // Cleaning up memory
  free(h_input);
  free(h_cpu_result);
  free(h_gpu_result);
  return 0;
}






